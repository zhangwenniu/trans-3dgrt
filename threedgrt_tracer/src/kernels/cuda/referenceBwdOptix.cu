#include "hip/hip_runtime.h"
// SPDX-FileCopyrightText: Copyright (c) 2025 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
// SPDX-License-Identifier: Apache-2.0
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
// http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <3dgrt/pipelineParameters.h>
#include <3dgrt/kernels/cuda/gaussianParticles.cuh>
// clang-format on

extern "C" {
__constant__ PipelineBackwardParameters params;
}

struct RayHit {
    unsigned int particleId;
    float distance;

    static constexpr unsigned int InvalidParticleId = 0xFFFFFFFF;
    static constexpr float InfiniteDistance         = 1e20f;
};
using RayPayload = RayHit[PipelineParameters::MaxNumHitPerTrace];

static __device__ __inline__ float2 intersectAABB(const OptixAabb& aabb, const float3& rayOri, const float3& rayDir) {
    const float3 t0   = (make_float3(aabb.minX, aabb.minY, aabb.minZ) - rayOri) / rayDir;
    const float3 t1   = (make_float3(aabb.maxX, aabb.maxY, aabb.maxZ) - rayOri) / rayDir;
    const float3 tmax = make_float3(fmaxf(t0.x, t1.x), fmaxf(t0.y, t1.y), fmaxf(t0.z, t1.z));
    const float3 tmin = make_float3(fminf(t0.x, t1.x), fminf(t0.y, t1.y), fminf(t0.z, t1.z));
    return float2{fmaxf(0.f, fmaxf(tmin.x, fmaxf(tmin.y, tmin.z))), fminf(tmax.x, fminf(tmax.y, tmax.z))};
}

static __device__ __inline__ uint32_t optixPrimitiveIndex() {
    return PipelineParameters::InstancePrimitive ? optixGetInstanceIndex() : (PipelineParameters::CustomPrimitive ? optixGetPrimitiveIndex() : static_cast<uint32_t>(optixGetPrimitiveIndex() / params.gPrimNumTri));
}

static __device__ __inline__ void trace(
    RayPayload& rayPayload,
    const float3& rayOri,
    const float3& rayDir,
    const float tmin,
    const float tmax) {
    uint32_t r0, r1, r2, r3, r4, r5, r6, r7, r8, r9, r10, r11, r12, r13, r14, r15,
        r16, r17, r18, r19, r20, r21, r22, r23, r24, r25, r26, r27, r28, r29, r30, r31;
    r0 = r2 = r4 = r6 = r8 = r10 = r12 = r14 = r16 = r18 = r20 = r22 = r24 = r26 = r28 = r30 = RayHit::InvalidParticleId;
    r1 = r3 = r5 = r7 = r9 = r11 = r13 = r15 = r17 = r19 = r21 = r23 = r25 = r27 = r29 = r31 = __float_as_int(RayHit::InfiniteDistance);

    // Trace the ray against our scene hierarchy
    optixTrace(params.handle, rayOri, rayDir,
               tmin,                     // Min intersection distance
               tmax,                     // Max intersection distance
               0.0f,                     // rayTime -- used for motion blur
               OptixVisibilityMask(255), // Specify always visible
               OPTIX_RAY_FLAG_DISABLE_CLOSESTHIT | (PipelineParameters::SurfelPrimitive ? OPTIX_RAY_FLAG_NONE : OPTIX_RAY_FLAG_CULL_BACK_FACING_TRIANGLES),
               0, // SBT offset   -- See SBT discussion
               1, // SBT stride   -- See SBT discussion
               0, // missSBTIndex -- See SBT discussion
               r0, r1, r2, r3, r4, r5, r6, r7, r8, r9, r10, r11, r12, r13, r14, r15,
               r16, r17, r18, r19, r20, r21, r22, r23, r24, r25, r26, r27, r28, r29, r30, r31);

    rayPayload[0].particleId  = r0;
    rayPayload[0].distance    = __uint_as_float(r1);
    rayPayload[1].particleId  = r2;
    rayPayload[1].distance    = __uint_as_float(r3);
    rayPayload[2].particleId  = r4;
    rayPayload[2].distance    = __uint_as_float(r5);
    rayPayload[3].particleId  = r6;
    rayPayload[3].distance    = __uint_as_float(r7);
    rayPayload[4].particleId  = r8;
    rayPayload[4].distance    = __uint_as_float(r9);
    rayPayload[5].particleId  = r10;
    rayPayload[5].distance    = __uint_as_float(r11);
    rayPayload[6].particleId  = r12;
    rayPayload[6].distance    = __uint_as_float(r13);
    rayPayload[7].particleId  = r14;
    rayPayload[7].distance    = __uint_as_float(r15);
    rayPayload[8].particleId  = r16;
    rayPayload[8].distance    = __uint_as_float(r17);
    rayPayload[9].particleId  = r18;
    rayPayload[9].distance    = __uint_as_float(r19);
    rayPayload[10].particleId = r20;
    rayPayload[10].distance   = __uint_as_float(r21);
    rayPayload[11].particleId = r22;
    rayPayload[11].distance   = __uint_as_float(r23);
    rayPayload[12].particleId = r24;
    rayPayload[12].distance   = __uint_as_float(r25);
    rayPayload[13].particleId = r26;
    rayPayload[13].distance   = __uint_as_float(r27);
    rayPayload[14].particleId = r28;
    rayPayload[14].distance   = __uint_as_float(r29);
    rayPayload[15].particleId = r30;
    rayPayload[15].distance   = __uint_as_float(r31);
}

extern "C" __global__ void __raygen__rg() {
    const uint3 idx = optixGetLaunchIndex();
    if ((idx.x > params.frameBounds.x) || (idx.y > params.frameBounds.y)) {
        return;
    }

    const float3 rayOrigin    = params.rayWorldOrigin(idx);
    const float3 rayDirection = params.rayWorldDirection(idx);

    float3 rayIntegratedRadiance     = make_float3(params.rayRadiance[idx.z][idx.y][idx.x][0], params.rayRadiance[idx.z][idx.y][idx.x][1], params.rayRadiance[idx.z][idx.y][idx.x][2]);
    float rayIntegratedTransmittance = 1.0f - params.rayDensity[idx.z][idx.y][idx.x][0];
    float rayIntegratedHitDistance   = params.rayHitDistance[idx.z][idx.y][idx.x][0];
    float rayMaxHitDistance          = params.rayHitDistance[idx.z][idx.y][idx.x][1];

    float3 rayRadianceGrad     = make_float3(params.rayRadianceGrad[idx.z][idx.y][idx.x][0], params.rayRadianceGrad[idx.z][idx.y][idx.x][1], params.rayRadianceGrad[idx.z][idx.y][idx.x][2]);
    float rayTransmittanceGrad = -1.0f * params.rayDensityGrad[idx.z][idx.y][idx.x][0];
    float rayHitDistanceGrad   = params.rayHitDistanceGrad[idx.z][idx.y][idx.x][0];

    constexpr float epsT = 1e-9;

    float2 minMaxT   = intersectAABB(params.aabb, rayOrigin, rayDirection);
    float startT     = fmaxf(0.0f, minMaxT.x - epsT);
    const float endT = fminf(rayMaxHitDistance, minMaxT.y) + epsT;

    float3 rayRadiance     = make_float3(0.f);
    float rayTransmittance = 1.f;
    float rayHitDistance   = 0.f;

    // 定义射线原点和方向的梯度变量
    float3 rayOriginGrad = make_float3(0.f);
    float3 rayDirectionGrad = make_float3(0.f);

    RayPayload rayPayload;

    while (startT < endT) {
        trace(rayPayload, rayOrigin, rayDirection, startT + epsT, endT);
        if (rayPayload[0].particleId == RayHit::InvalidParticleId) {
            break;
        }

#pragma unroll
        for (int i = 0; i < PipelineParameters::MaxNumHitPerTrace; i++) {
            const RayHit rayHit = rayPayload[i];

            if (rayHit.particleId != RayHit::InvalidParticleId) {
                processHitBwd<PipelineParameters::ParticleKernelDegree, PipelineParameters::SurfelPrimitive>(
                    rayOrigin,
                    rayDirection,
                    rayHit.particleId,
                    params.particleDensity,
                    params.particleDensityGrad,
                    params.particleRadiance,
                    params.particleRadianceGrad,
                    params.hitMinGaussianResponse,
                    params.alphaMinThreshold,
                    params.minTransmittance,
                    params.sphDegree,
                    rayIntegratedTransmittance,
                    rayTransmittance,
                    rayTransmittanceGrad,
                    rayIntegratedRadiance,
                    rayRadiance,
                    rayRadianceGrad,
                    rayIntegratedHitDistance,
                    rayHitDistance,
                    rayHitDistanceGrad,
                    &rayOriginGrad,      // 新增：传入射线原点梯度
                    &rayDirectionGrad);  // 新增：传入射线方向梯度

                startT = fmaxf(startT, rayHit.distance);
            }
        }
    }

    // 循环结束后写入梯度
    if (params.rayOriginGrad.size(0) > 0 && 
        idx.x < params.rayOriginGrad.size(2) && 
        idx.y < params.rayOriginGrad.size(1) && 
        idx.z < params.rayOriginGrad.size(0)) {
        
        params.rayOriginGrad[idx.z][idx.y][idx.x][0] = rayOriginGrad.x;
        params.rayOriginGrad[idx.z][idx.y][idx.x][1] = rayOriginGrad.y;
        params.rayOriginGrad[idx.z][idx.y][idx.x][2] = rayOriginGrad.z;
    }
    
    if (params.rayDirectionGrad.size(0) > 0 && 
        idx.x < params.rayDirectionGrad.size(2) && 
        idx.y < params.rayDirectionGrad.size(1) && 
        idx.z < params.rayDirectionGrad.size(0)) {
        
        params.rayDirectionGrad[idx.z][idx.y][idx.x][0] = rayDirectionGrad.x;
        params.rayDirectionGrad[idx.z][idx.y][idx.x][1] = rayDirectionGrad.y;
        params.rayDirectionGrad[idx.z][idx.y][idx.x][2] = rayDirectionGrad.z;
    }
}

extern "C" __global__ void __intersection__is() {
    float hitDistance;
    const bool intersect = PipelineParameters::InstancePrimitive ? intersectInstanceParticle(optixGetObjectRayOrigin(),
                                                                                             optixGetObjectRayDirection(),
                                                                                             optixGetInstanceIndex(),
                                                                                             optixGetRayTmin(),
                                                                                             optixGetRayTmax(),
                                                                                             params.hitMaxParticleSquaredDistance,
                                                                                             hitDistance)
                                                                 : intersectCustomParticle(optixGetWorldRayOrigin(),
                                                                                           optixGetWorldRayDirection(),
                                                                                           optixGetPrimitiveIndex(),
                                                                                           params.particleDensity,
                                                                                           optixGetRayTmin(),
                                                                                           optixGetRayTmax(),
                                                                                           params.hitMaxParticleSquaredDistance,
                                                                                           hitDistance);
    if (intersect) {
        optixReportIntersection(hitDistance, 0);
    }
}

#define compareAndSwapHitPayloadValue(hit, i_id, i_distance)                      \
    {                                                                             \
        const float distance = __uint_as_float(optixGetPayload_##i_distance##()); \
        if (hit.distance < distance) {                                            \
            optixSetPayload_##i_distance##(__float_as_uint(hit.distance));        \
            const uint32_t id = optixGetPayload_##i_id##();                       \
            optixSetPayload_##i_id##(hit.particleId);                             \
            hit.distance   = distance;                                            \
            hit.particleId = id;                                                  \
        }                                                                         \
    }

extern "C" __global__ void __anyhit__ah() {
    RayHit hit = RayHit{optixPrimitiveIndex(), optixGetRayTmax()};

    if (hit.distance < __uint_as_float(optixGetPayload_31())) {
        compareAndSwapHitPayloadValue(hit, 0, 1);
        compareAndSwapHitPayloadValue(hit, 2, 3);
        compareAndSwapHitPayloadValue(hit, 4, 5);
        compareAndSwapHitPayloadValue(hit, 6, 7);
        compareAndSwapHitPayloadValue(hit, 8, 9);
        compareAndSwapHitPayloadValue(hit, 10, 11);
        compareAndSwapHitPayloadValue(hit, 12, 13);
        compareAndSwapHitPayloadValue(hit, 14, 15);
        compareAndSwapHitPayloadValue(hit, 16, 17);
        compareAndSwapHitPayloadValue(hit, 18, 19);
        compareAndSwapHitPayloadValue(hit, 20, 21);
        compareAndSwapHitPayloadValue(hit, 22, 23);
        compareAndSwapHitPayloadValue(hit, 24, 25);
        compareAndSwapHitPayloadValue(hit, 26, 27);
        compareAndSwapHitPayloadValue(hit, 28, 29);
        compareAndSwapHitPayloadValue(hit, 30, 31);

        // ignore all inserted hits, expect if the last one
        if (__uint_as_float(optixGetPayload_31()) > optixGetRayTmax()) {
            optixIgnoreIntersection();
        }
    }
}
